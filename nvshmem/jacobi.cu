#include "hip/hip_runtime.h"
/* Copyright (c) 2024 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include <mpi.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstdio>
#include <iostream>
#include <sstream>
#include <cstdlib>

#ifdef HAVE_CUB
#include <cub/block/block_reduce.cuh>
#endif  // HAVE_CUB

#define MPI_CALL(call)                                                                \
    {                                                                                 \
        int mpi_status = call;                                                        \
        if (MPI_SUCCESS != mpi_status) {                                              \
            char mpi_error_string[MPI_MAX_ERROR_STRING];                              \
            int mpi_error_string_length = 0;                                          \
            MPI_Error_string(mpi_status, mpi_error_string, &mpi_error_string_length); \
            if (NULL != mpi_error_string)                                             \
                fprintf(stderr,                                                       \
                        "ERROR: MPI call \"%s\" in line %d of file %s failed "        \
                        "with %s "                                                    \
                        "(%d).\n",                                                    \
                        #call, __LINE__, __FILE__, mpi_error_string, mpi_status);     \
            else                                                                      \
                fprintf(stderr,                                                       \
                        "ERROR: MPI call \"%s\" in line %d of file %s failed "        \
                        "with %d.\n",                                                 \
                        #call, __LINE__, __FILE__, mpi_status);                       \
            exit( mpi_status );                                                       \
        }                                                                             \
    }

#ifdef USE_NVTX
#include <nvtx3/nvToolsExt.h>

const uint32_t colors[] = {0x0000ff00, 0x000000ff, 0x00ffff00, 0x00ff00ff,
                           0x0000ffff, 0x00ff0000, 0x00ffffff};
const int num_colors = sizeof(colors) / sizeof(uint32_t);

#define PUSH_RANGE(name, cid)                              \
    {                                                      \
        int color_id = cid;                                \
        color_id = color_id % num_colors;                  \
        nvtxEventAttributes_t eventAttrib = {0};           \
        eventAttrib.version = NVTX_VERSION;                \
        eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;  \
        eventAttrib.colorType = NVTX_COLOR_ARGB;           \
        eventAttrib.color = colors[color_id];              \
        eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
        eventAttrib.message.ascii = name;                  \
        nvtxRangePushEx(&eventAttrib);                     \
    }
#define POP_RANGE nvtxRangePop();
#else
#define PUSH_RANGE(name, cid)
#define POP_RANGE
#endif

#define CUDA_RT_CALL(call)                                                                  \
    {                                                                                       \
        hipError_t cudaStatus = call;                                                      \
        if (hipSuccess != cudaStatus) {                                                    \
            fprintf(stderr,                                                                 \
                    "ERROR: CUDA RT call \"%s\" in line %d of file %s failed "              \
                    "with "                                                                 \
                    "%s (%d).\n",                                                           \
                    #call, __LINE__, __FILE__, hipGetErrorString(cudaStatus), cudaStatus); \
            exit( cudaStatus );                                                             \
        }                                                                                   \
    }

// convert NVSHMEM_SYMMETRIC_SIZE string to long long unsigned int
long long unsigned int parse_nvshmem_symmetric_size(char *value) {
    long long unsigned int units, size;

    assert(value != NULL);

    if (strchr(value, 'G') != NULL) {
        units=1e9;
    } else if (strchr(value, 'M') != NULL) {
        units=1e6;
    } else if (strchr(value, 'K') != NULL) {
        units=1e3;
    } else {
        units=1;
    }

    assert(atof(value) >= 0);
    size = (long long unsigned int) atof(value) * units;

    return size;
}

typedef float real;
constexpr real tol = 1.0e-8;

const real PI = 2.0 * std::asin(1.0);

/* This kernel implements neighborhood synchronization for Jacobi. It updates
   the neighbor PEs about its arrival and waits for notification from them. */
__global__ void syncneighborhood_kernel(int my_pe, int num_pes, uint64_t* sync_arr,
                                        long counter) {
    int next_rank = (my_pe + 1) % num_pes;
    int prev_rank = (my_pe == 0) ? num_pes - 1 : my_pe - 1;
    nvshmem_quiet(); /* To ensure all prior nvshmem operations have been completed */

    /* Notify neighbors about arrival */
    nvshmemx_signal_op(sync_arr, counter, NVSHMEM_SIGNAL_SET, next_rank);
    nvshmemx_signal_op(sync_arr + 1, counter, NVSHMEM_SIGNAL_SET, prev_rank);

    /* Wait for neighbors notification */
    nvshmem_uint64_wait_until_all(sync_arr, 2, NULL, NVSHMEM_CMP_GE, counter);
}

__global__ void initialize_boundaries(real* __restrict__ const a_new, real* __restrict__ const a,
                                      const real pi, const int offset, const int nx,
                                      const int my_ny, int ny) {
    for (int iy = blockIdx.x * blockDim.x + threadIdx.x; iy < my_ny; iy += blockDim.x * gridDim.x) {
        const real y0 = sin(2.0 * pi * (offset + iy) / (ny - 1));
        a[(iy + 1) * nx + 0] = y0;
        a[(iy + 1) * nx + (nx - 1)] = y0;
        a_new[(iy + 1) * nx + 0] = y0;
        a_new[(iy + 1) * nx + (nx - 1)] = y0;
    }
}

template <int BLOCK_DIM_X, int BLOCK_DIM_Y>
__global__ void jacobi_kernel(real* __restrict__ const a_new, const real* __restrict__ const a,
                              real* __restrict__ const l2_norm, const int iy_start,
                              const int iy_end, const int nx, const int top_pe, const int top_iy,
                              const int bottom_pe, const int bottom_iy) {
#ifdef HAVE_CUB
    typedef hipcub::BlockReduce<real, BLOCK_DIM_X, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, BLOCK_DIM_Y>
        BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
#endif  // HAVE_CUB
    int iy = blockIdx.y * blockDim.y + threadIdx.y + iy_start;
    int ix = blockIdx.x * blockDim.x + threadIdx.x + 1;
    real local_l2_norm = 0.0;

    if (iy < iy_end && ix < (nx - 1)) {
        const real new_val = 0.25 * (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] +
                                     a[(iy + 1) * nx + ix] + a[(iy - 1) * nx + ix]);
        a_new[iy * nx + ix] = new_val;

        real residue = new_val - a[iy * nx + ix];
        local_l2_norm += residue * residue;

        if (iy_start == iy) {
            nvshmem_float_p(a_new + top_iy * nx + ix, new_val, top_pe);
        }
        if ((iy_end - 1) == iy) {
            nvshmem_float_p(a_new + bottom_iy * nx + ix, new_val, bottom_pe);
        }
    }
#ifdef HAVE_CUB
    real block_l2_norm = BlockReduce(temp_storage).Sum(local_l2_norm);
    if (0 == threadIdx.y && 0 == threadIdx.x) atomicAdd(l2_norm, block_l2_norm);
#else
    atomicAdd(l2_norm, local_l2_norm);
#endif  // HAVE_CUB
}

template <int BLOCK_DIM_X, int BLOCK_DIM_Y>
__global__ void jacobi_block_comm_kernel(real* __restrict__ const a_new, const real* __restrict__ const a,
                              real* __restrict__ const l2_norm, const int iy_start,
                              const int iy_end, const int nx, const int top_pe, const int top_iy,
                              const int bottom_pe, const int bottom_iy) {
#ifdef HAVE_CUB
    typedef hipcub::BlockReduce<real, BLOCK_DIM_X, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, BLOCK_DIM_Y>
        BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
#endif  // HAVE_CUB
    int iy = blockIdx.y * blockDim.y + threadIdx.y + iy_start;
    int ix = blockIdx.x * blockDim.x + threadIdx.x + 1;
    real local_l2_norm = 0.0;

    if (iy < iy_end && ix < (nx - 1)) {
        const real new_val = 0.25 * (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] +
                                     a[(iy + 1) * nx + ix] + a[(iy - 1) * nx + ix]);
        a_new[iy * nx + ix] = new_val;
        real residue = new_val - a[iy * nx + ix];
        local_l2_norm += residue * residue;
    }

    /* starting (x, y) coordinate of the block */
    int block_iy =
        iy - threadIdx.y; /* Alternatively, block_iy = blockIdx.y * blockDim.y + iy_start */
    int block_ix = ix - threadIdx.x; /* Alternatively, block_ix = blockIdx.x * blockDim.x + 1 */

    /* Communicate the boundaries */
    if ((block_iy <= iy_start) && (iy_start < block_iy + blockDim.y)) {
        nvshmemx_float_put_nbi_block(a_new + top_iy * nx + block_ix, a_new + iy_start * nx + block_ix,
                                   min(blockDim.x, nx - 1 - block_ix), top_pe);
    }
    if ((block_iy < iy_end) && (iy_end <= block_iy + blockDim.y)) {
        nvshmemx_float_put_nbi_block(a_new + bottom_iy * nx + block_ix,
                                   a_new + (iy_end - 1) * nx + block_ix,
                                   min(blockDim.x, nx - 1 - block_ix), bottom_pe);
    }

#ifdef HAVE_CUB
    real block_l2_norm = BlockReduce(temp_storage).Sum(local_l2_norm);
    if (0 == threadIdx.y && 0 == threadIdx.x) atomicAdd(l2_norm, block_l2_norm);
#else
    atomicAdd(l2_norm, local_l2_norm);
#endif  // HAVE_CUB
}

double single_gpu(const int nx, const int ny, const int iter_max, real* const a_ref_h,
                  const int nccheck, const bool print, int mype);

template <typename T>
T get_argval(char** begin, char** end, const std::string& arg, const T default_val) {
    T argval = default_val;
    char** itr = std::find(begin, end, arg);
    if (itr != end && ++itr != end) {
        std::istringstream inbuf(*itr);
        inbuf >> argval;
    }
    return argval;
}

bool get_arg(char** begin, char** end, const std::string& arg) {
    char** itr = std::find(begin, end, arg);
    if (itr != end) {
        return true;
    }
    return false;
}

struct l2_norm_buf {
    hipEvent_t copy_done;
    real* d;
    real* h;
};

int main(int argc, char* argv[]) {
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 1000);
    const int nx = get_argval<int>(argv, argv + argc, "-nx", 16384);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", 16384);
    const int nccheck = get_argval<int>(argv, argv + argc, "-nccheck", 1);
    const bool csv = get_arg(argv, argv + argc, "-csv");
    const bool use_block_comm = get_arg(argv, argv + argc, "-use_block_comm");
    const bool norm_overlap = get_arg(argv, argv + argc, "-norm_overlap");
    const bool neighborhood_sync = get_arg(argv, argv + argc, "-neighborhood_sync");

    if (nccheck != 1) {
        fprintf(stderr, "Only nccheck=1 is supported\n");
        return -1;
    }

    real* a_new;

    real* a_ref_h;
    real* a_h;
    double runtime_serial = 0.0;

    real l2_norms[2];

    int rank = 0, size = 1;
    MPI_CALL(MPI_Init(&argc, &argv));
    MPI_CALL(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
    MPI_CALL(MPI_Comm_size(MPI_COMM_WORLD, &size));

    int num_devices;
    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));

    int local_rank = -1;
    int local_size = 1;
    {
        MPI_Comm local_comm;
        MPI_CALL(MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank, MPI_INFO_NULL,
                                     &local_comm));

        MPI_CALL(MPI_Comm_rank(local_comm, &local_rank));
        MPI_CALL(MPI_Comm_size(local_comm, &local_size));

        MPI_CALL(MPI_Comm_free(&local_comm));
    }
    if ( 1 < num_devices && num_devices < local_size )
    {
        fprintf(stderr,"ERROR Number of visible devices (%d) is less than number of ranks on the node (%d)!\n", num_devices, local_size);
        MPI_CALL(MPI_Finalize());
        return 1;
    }
    if ( 1 == num_devices ) {
        // Only 1 device visible, assuming GPU affinity is handled via CUDA_VISIBLE_DEVICES
        CUDA_RT_CALL(hipSetDevice(0));
    } else {
        CUDA_RT_CALL(hipSetDevice(local_rank));
    }
    CUDA_RT_CALL(hipFree(0));

    MPI_Comm mpi_comm;
    nvshmemx_init_attr_t attr;

    mpi_comm = MPI_COMM_WORLD;
    attr.mpi_comm = &mpi_comm;
    // Set symmetric heap size for nvshmem based on problem size
    // Its default value in nvshmem is 1 GB which is not sufficient
    // for large mesh sizes
    long long unsigned int mesh_size_per_rank = nx * (((ny - 2) + size - 1) / size + 2);
    long long unsigned int required_symmetric_heap_size =
        2 * mesh_size_per_rank * sizeof(real) *
        1.1;  // Factor 2 is because 2 arrays are allocated - a and a_new
              // 1.1 factor is just for alignment or other usage

    char * value = getenv("NVSHMEM_SYMMETRIC_SIZE");
    if (value) { /* env variable is set */
        long long unsigned int size_env = parse_nvshmem_symmetric_size(value);
        if (size_env < required_symmetric_heap_size) {
            fprintf(stderr, "ERROR: Minimum NVSHMEM_SYMMETRIC_SIZE = %lluB, Current NVSHMEM_SYMMETRIC_SIZE=%s\n", required_symmetric_heap_size, value);
            MPI_CALL(MPI_Finalize());
            return -1;
        }
    } else {
        char symmetric_heap_size_str[100];
        sprintf(symmetric_heap_size_str, "%llu", required_symmetric_heap_size);
        if (!rank && !csv)
            printf("Setting environment variable NVSHMEM_SYMMETRIC_SIZE = %llu\n", required_symmetric_heap_size);
        setenv("NVSHMEM_SYMMETRIC_SIZE", symmetric_heap_size_str, 1);
    }
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);

    int npes = nvshmem_n_pes();
    int mype = nvshmem_my_pe();

    nvshmem_barrier_all();

    bool result_correct = true;
    real* a;

    hipStream_t compute_stream;
    hipStream_t reset_l2_norm_stream;
    hipEvent_t compute_done[2];
    hipEvent_t reset_l2_norm_done[2];

    l2_norm_buf l2_norm_bufs[2];

    CUDA_RT_CALL(hipHostMalloc(&a_ref_h, nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipHostMalloc(&a_h, nx * ny * sizeof(real)));
    runtime_serial = single_gpu(nx, ny, iter_max, a_ref_h, nccheck, !csv && (0 == mype), mype);

    nvshmem_barrier_all();

    // ny - 2 rows are distributed amongst `size` ranks in such a way
    // that each rank gets either (ny - 2) / size or (ny - 2) / size + 1 rows.
    // This optimizes load balancing when (ny - 2) % size != 0
    int chunk_size;
    int chunk_size_low = (ny - 2) / npes;
    int chunk_size_high = chunk_size_low + 1;
    // To calculate the number of ranks that need to compute an extra row,
    // the following formula is derived from this equation:
    // num_ranks_low * chunk_size_low + (size - num_ranks_low) * (chunk_size_low + 1) = ny - 2
    int num_ranks_low = npes * chunk_size_low + npes -
                        (ny - 2);  // Number of ranks with chunk_size = chunk_size_low
    if (mype < num_ranks_low)
        chunk_size = chunk_size_low;
    else
        chunk_size = chunk_size_high;

    a = (real*)nvshmem_malloc(
        nx * (chunk_size_high + 2) *
        sizeof(real));  // Using chunk_size_high so that it is same across all PEs
    a_new = (real*)nvshmem_malloc(nx * (chunk_size_high + 2) * sizeof(real));

    hipMemset(a, 0, nx * (chunk_size + 2) * sizeof(real));
    hipMemset(a_new, 0, nx * (chunk_size + 2) * sizeof(real));

    // Calculate local domain boundaries
    int iy_start_global;  // My start index in the global array
    if (mype < num_ranks_low) {
        iy_start_global = mype * chunk_size_low + 1;
    } else {
        iy_start_global =
            num_ranks_low * chunk_size_low + (mype - num_ranks_low) * chunk_size_high + 1;
    }
    int iy_end_global = iy_start_global + chunk_size - 1;  // My last index in the global array
    // do not process boundaries
    iy_end_global = std::min(iy_end_global, ny - 4);

    int iy_start = 1;
    int iy_end = (iy_end_global - iy_start_global + 1) + iy_start;

    // calculate boundary indices for top and bottom boundaries
    int top_pe = mype > 0 ? mype - 1 : (npes - 1);
    int bottom_pe = (mype + 1) % npes;

    int iy_end_top = (top_pe < num_ranks_low) ? chunk_size_low + 1 : chunk_size_high + 1;
    int iy_start_bottom = 0;

    // Set diriclet boundary conditions on left and right boundary
    initialize_boundaries<<<(ny / npes) / 128 + 1, 128>>>(a, a_new, PI, iy_start_global - 1, nx,
                                                          chunk_size, ny - 2);
    CUDA_RT_CALL(hipGetLastError());
    CUDA_RT_CALL(hipDeviceSynchronize());

    CUDA_RT_CALL(hipStreamCreateWithFlags(&compute_stream, hipStreamNonBlocking));
    CUDA_RT_CALL(hipStreamCreate(&reset_l2_norm_stream));
    CUDA_RT_CALL(hipEventCreateWithFlags(&compute_done[0], hipEventDisableTiming));
    CUDA_RT_CALL(hipEventCreateWithFlags(&compute_done[1], hipEventDisableTiming));
    CUDA_RT_CALL(hipEventCreateWithFlags(&reset_l2_norm_done[0], hipEventDisableTiming));
    CUDA_RT_CALL(hipEventCreateWithFlags(&reset_l2_norm_done[1], hipEventDisableTiming));

    for (int i = 0; i < 2; ++i) {
        CUDA_RT_CALL(hipEventCreateWithFlags(&l2_norm_bufs[i].copy_done, hipEventDisableTiming));
        CUDA_RT_CALL(hipMalloc(&l2_norm_bufs[i].d, sizeof(real)));
        CUDA_RT_CALL(hipMemset(l2_norm_bufs[i].d, 0, sizeof(real)));
        CUDA_RT_CALL(hipHostMalloc(&l2_norm_bufs[i].h, sizeof(real)));
        *(l2_norm_bufs[i].h) = 1.0;
    }

    nvshmemx_barrier_all_on_stream(compute_stream);
    MPI_CALL(MPI_Allreduce(l2_norm_bufs[0].h, &l2_norms[0], 1, MPI_FLOAT, MPI_SUM, MPI_COMM_WORLD));
    MPI_CALL(MPI_Allreduce(l2_norm_bufs[1].h, &l2_norms[1], 1, MPI_FLOAT, MPI_SUM, MPI_COMM_WORLD));
    CUDA_RT_CALL(hipDeviceSynchronize());

    if (!mype) {
        if (!csv) printf("Jacobi relaxation: %d iterations on %d x %d mesh\n", iter_max, ny, nx);
    }

    constexpr int dim_block_x = 32;
    constexpr int dim_block_y = 32;
    dim3 dim_grid((nx + dim_block_x - 1) / dim_block_x,
                  (chunk_size + dim_block_y - 1) / dim_block_y, 1);
    constexpr int dim_block_block_comm_x = 1024;
    constexpr int dim_block_block_comm_y = 1;
    dim3 dim_grid_block_comm((nx + dim_block_block_comm_x - 1) / dim_block_block_comm_x,
                  (chunk_size + dim_block_block_comm_y - 1) / dim_block_block_comm_y, 1);

    int iter = 0;
    if (!mype) {
        for (int i = 0; i < 2; ++i) {
            l2_norms[i] = 1.0;
        }
    }

    /* Used by syncneighborhood kernel */
    uint64_t* sync_arr = NULL;
    sync_arr = (uint64_t*)nvshmem_malloc(2 * sizeof(uint64_t));
    hipMemsetAsync(sync_arr, 0, 2 * sizeof(uint64_t), compute_stream);
    hipStreamSynchronize(compute_stream);
    long synccounter = 1;

    CUDA_RT_CALL(hipDeviceSynchronize());
    MPI_CALL(MPI_Barrier(MPI_COMM_WORLD));

    double start = MPI_Wtime();
    PUSH_RANGE("Jacobi solve", 0)
    bool l2_norm_greater_than_tol = true;
    
    while (l2_norm_greater_than_tol && iter < iter_max) {
        int prev = 0;
        int curr = 0;
        if ( norm_overlap ) {
            // on new iteration: old current vars are now previous vars, old
            // previous vars are no longer needed
            prev = iter % 2;
            curr = (iter + 1) % 2;
        }

        CUDA_RT_CALL(hipStreamWaitEvent(compute_stream, reset_l2_norm_done[curr], 0));
        if (use_block_comm) {
            jacobi_block_comm_kernel<dim_block_block_comm_x, dim_block_block_comm_y>
                <<<dim_grid_block_comm, {dim_block_block_comm_x, dim_block_block_comm_y, 1}, 0, compute_stream>>>(
                    a_new, a, l2_norm_bufs[curr].d, iy_start, iy_end, nx, top_pe, iy_end_top, bottom_pe,
                    iy_start_bottom);
        } else {
            jacobi_kernel<dim_block_x, dim_block_y>
                <<<dim_grid, {dim_block_x, dim_block_y, 1}, 0, compute_stream>>>(
                    a_new, a, l2_norm_bufs[curr].d, iy_start, iy_end, nx, top_pe, iy_end_top, bottom_pe,
                    iy_start_bottom);
        }
        CUDA_RT_CALL(hipGetLastError());

        if ( neighborhood_sync ) {
            /* Instead of using nvshmemx_barrier_all_on_stream, we are using a custom implementation
            of barrier that just synchronizes with the neighbor PEs that is the PEs with whom a PE
            communicates. This will perform faster than a global barrier that would do redundant
            synchronization for this application. */
            syncneighborhood_kernel<<<1, 1, 0, compute_stream>>>(mype, npes, sync_arr, synccounter);
            CUDA_RT_CALL(hipGetLastError());
            synccounter++;
        } else {
            nvshmemx_barrier_all_on_stream(compute_stream);
        }

        // perform L2 norm calculation
        if ((iter % nccheck) == 0 || (!csv && (iter % 100) == 0)) {
            // as soon as computation is complete -> D2H-copy L2 norm
            CUDA_RT_CALL(hipMemcpyAsync(l2_norm_bufs[curr].h, l2_norm_bufs[curr].d, sizeof(real),
                                         hipMemcpyDeviceToHost, compute_stream));
            CUDA_RT_CALL(hipEventRecord(l2_norm_bufs[curr].copy_done, compute_stream));

            // ensure previous D2H-copy is completed before using the data for
            // calculation
            CUDA_RT_CALL(hipEventSynchronize(l2_norm_bufs[prev].copy_done));

            MPI_CALL(MPI_Allreduce(l2_norm_bufs[prev].h, &l2_norms[prev], 1, MPI_FLOAT, MPI_SUM,
                                   MPI_COMM_WORLD));

            l2_norms[prev] = std::sqrt(l2_norms[prev]);
            l2_norm_greater_than_tol = (l2_norms[prev] > tol);

            if (!csv && (iter % 100) == 0) {
                if (!mype) printf("%5d, %0.6f\n", iter, l2_norms[prev]);
            }

            // reset everything for next iteration
            l2_norms[prev] = 0.0;
            *(l2_norm_bufs[prev].h) = 0.0;
            CUDA_RT_CALL(hipMemcpyAsync(l2_norm_bufs[prev].d, l2_norm_bufs[prev].h, sizeof(real),
                                         hipMemcpyHostToDevice, reset_l2_norm_stream));
            CUDA_RT_CALL(hipEventRecord(reset_l2_norm_done[prev], reset_l2_norm_stream));
        }

        std::swap(a_new, a);
        iter++;
    }

    CUDA_RT_CALL(hipDeviceSynchronize());
    MPI_CALL(MPI_Barrier(MPI_COMM_WORLD));
    double stop = MPI_Wtime();
    POP_RANGE

    nvshmem_barrier_all();

    CUDA_RT_CALL(hipMemcpy(a_h + iy_start_global * nx, a + nx,
                            std::min(ny - 2 - iy_start_global, chunk_size) * nx * sizeof(real),
                            hipMemcpyDeviceToHost));

    result_correct = true;
    for (int iy = iy_start_global; result_correct && (iy < iy_end_global); ++iy) {
        for (int ix = 1; result_correct && (ix < (nx - 1)); ++ix) {
            if (std::fabs(a_ref_h[iy * nx + ix] - a_h[iy * nx + ix]) > tol) {
                fprintf(stderr,
                        "ERROR on rank %d: a[%d * %d + %d] = %f does not match %f "
                        "(reference)\n",
                        rank, iy, nx, ix, a_h[iy * nx + ix], a_ref_h[iy * nx + ix]);
                result_correct = false;
            }
        }
    }

    int global_result_correct = 1;
    MPI_CALL(MPI_Allreduce(&result_correct, &global_result_correct, 1, MPI_INT, MPI_MIN,
                           MPI_COMM_WORLD));
    result_correct = global_result_correct;

    if (!mype && result_correct) {
        if (csv) {
            printf("nvshmem");
            if (use_block_comm)
                printf("-use_block_comm");
            if (norm_overlap)
                printf("-norm_overlap");
            if (neighborhood_sync)
                printf("-neighborhood_sync");
            printf(", %d, %d, %d, %d, %d, 1, %f, %f\n", nx, ny, iter_max, nccheck, npes,
                   (stop - start), runtime_serial);
        } else {
            printf("Num GPUs: %d.\n", npes);
            printf(
                "%dx%d: 1 GPU: %8.4f s, %d GPUs: %8.4f s, speedup: %8.2f, "
                "efficiency: %8.2f \n",
                ny, nx, runtime_serial, npes, (stop - start), runtime_serial / (stop - start),
                runtime_serial / (npes * (stop - start)) * 100);
        }
    }

    for (int i = 0; i < 2; ++i) {
        CUDA_RT_CALL(hipHostFree(l2_norm_bufs[i].h));
        CUDA_RT_CALL(hipFree(l2_norm_bufs[i].d));
        CUDA_RT_CALL(hipEventDestroy(l2_norm_bufs[i].copy_done));
    }

    nvshmem_free(a);
    nvshmem_free(a_new);
    nvshmem_free(sync_arr);

    CUDA_RT_CALL(hipEventDestroy(reset_l2_norm_done[1]));
    CUDA_RT_CALL(hipEventDestroy(reset_l2_norm_done[0]));
    CUDA_RT_CALL(hipEventDestroy(compute_done[1]));
    CUDA_RT_CALL(hipEventDestroy(compute_done[0]));
    CUDA_RT_CALL(hipStreamDestroy(reset_l2_norm_stream));
    CUDA_RT_CALL(hipStreamDestroy(compute_stream));

    CUDA_RT_CALL(hipHostFree(a_h));
    CUDA_RT_CALL(hipHostFree(a_ref_h));

    nvshmem_finalize();
    MPI_CALL(MPI_Finalize());

    return (result_correct == 1) ? 0 : 1;
}

double single_gpu(const int nx, const int ny, const int iter_max, real* const a_ref_h,
                  const int nccheck, const bool print, int mype) {
    real* a;
    real* a_new;

    hipStream_t compute_stream;

    real* l2_norm_d;
    real* l2_norm_h;

    int iy_start = 1;
    int iy_end = ny - 3;

    CUDA_RT_CALL(hipMalloc((void**)&a, nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMalloc((void**)&a_new, nx * ny * sizeof(real)));

    CUDA_RT_CALL(hipMemset(a, 0, nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMemset(a_new, 0, nx * ny * sizeof(real)));

    // Set diriclet boundary conditions on left and right boarder
    initialize_boundaries<<<ny / 128 + 1, 128>>>(a, a_new, PI, 0, nx, ny - 2, ny - 2);

    CUDA_RT_CALL(hipGetLastError());
    CUDA_RT_CALL(hipDeviceSynchronize());

    CUDA_RT_CALL(hipStreamCreate(&compute_stream));

    CUDA_RT_CALL(hipMalloc(&l2_norm_d, sizeof(real)));
    CUDA_RT_CALL(hipHostMalloc(&l2_norm_h, sizeof(real)));

    CUDA_RT_CALL(hipDeviceSynchronize());

    if (print)
        printf(
            "Single GPU jacobi relaxation: %d iterations on %d x %d mesh with "
            "norm "
            "check every %d iterations\n",
            iter_max, ny, nx, nccheck);

    constexpr int dim_block_x = 32;
    constexpr int dim_block_y = 32;
    dim3 dim_grid((nx + dim_block_x - 1) / dim_block_x, ((ny - 2) + dim_block_y - 1) / dim_block_y,
                  1);

    int iter = 0;
    real l2_norm = 1.0;

    CUDA_RT_CALL(hipDeviceSynchronize());
    double start = MPI_Wtime();
    PUSH_RANGE("Jacobi solve", 0)

    while (l2_norm > tol && iter < iter_max) {
        CUDA_RT_CALL(hipMemsetAsync(l2_norm_d, 0, sizeof(real), compute_stream));

        jacobi_kernel<dim_block_x, dim_block_y>
            <<<dim_grid, {dim_block_x, dim_block_y, 1}, 0, compute_stream>>>(
                a_new, a, l2_norm_d, iy_start, iy_end, nx, mype, iy_end + 1, mype, (iy_start - 1));
        CUDA_RT_CALL(hipGetLastError());

        if ((iter % nccheck) == 0 || (print && ((iter % 100) == 0))) {
            CUDA_RT_CALL(hipMemcpyAsync(l2_norm_h, l2_norm_d, sizeof(real), hipMemcpyDeviceToHost,
                                         compute_stream));
            CUDA_RT_CALL(hipStreamSynchronize(compute_stream));
            l2_norm = *l2_norm_h;
            l2_norm = std::sqrt(l2_norm);
            if (print && (iter % 100) == 0) printf("%5d, %0.6f\n", iter, l2_norm);
        }

        std::swap(a_new, a);
        iter++;
    }
    CUDA_RT_CALL(hipDeviceSynchronize());
    POP_RANGE
    double stop = MPI_Wtime();

    CUDA_RT_CALL(hipMemcpy(a_ref_h, a, nx * ny * sizeof(real), hipMemcpyDeviceToHost));

    CUDA_RT_CALL(hipStreamDestroy(compute_stream));

    CUDA_RT_CALL(hipHostFree(l2_norm_h));
    CUDA_RT_CALL(hipFree(l2_norm_d));

    CUDA_RT_CALL(hipFree(a_new));
    CUDA_RT_CALL(hipFree(a));
    return (stop - start);
}
