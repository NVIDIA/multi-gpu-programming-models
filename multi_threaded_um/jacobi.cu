#include "hip/hip_runtime.h"
/* Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include <algorithm>
#include <cmath>
#include <cstdio>
#include <iostream>
#include <sstream>
#include <cstdlib>

#include <omp.h>

#ifdef HAVE_CUB
#include <cub/block/block_reduce.cuh>
#endif  // HAVE_CUB

#ifdef USE_NVTX
#include <nvtx3/nvToolsExt.h>

const uint32_t colors[] = {0x0000ff00, 0x000000ff, 0x00ffff00, 0x00ff00ff,
                           0x0000ffff, 0x00ff0000, 0x00ffffff};
const int num_colors = sizeof(colors) / sizeof(uint32_t);

#define PUSH_RANGE(name, cid)                              \
    {                                                      \
        int color_id = cid;                                \
        color_id = color_id % num_colors;                  \
        nvtxEventAttributes_t eventAttrib = {0};           \
        eventAttrib.version = NVTX_VERSION;                \
        eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;  \
        eventAttrib.colorType = NVTX_COLOR_ARGB;           \
        eventAttrib.color = colors[color_id];              \
        eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
        eventAttrib.message.ascii = name;                  \
        nvtxRangePushEx(&eventAttrib);                     \
    }
#define POP_RANGE nvtxRangePop();
#else
#define PUSH_RANGE(name, cid)
#define POP_RANGE
#endif

#define CUDA_RT_CALL(call)                                                                  \
    {                                                                                       \
        hipError_t cudaStatus = call;                                                      \
        if (hipSuccess != cudaStatus) {                                                    \
            fprintf(stderr,                                                                 \
                    "ERROR: CUDA RT call \"%s\" in line %d of file %s failed "              \
                    "with "                                                                 \
                    "%s (%d).\n",                                                           \
                    #call, __LINE__, __FILE__, hipGetErrorString(cudaStatus), cudaStatus); \
            exit( cudaStatus );                                                             \
        }                                                                                   \
    }

constexpr int MAX_NUM_DEVICES = 32;

typedef float real;
constexpr real tol = 1.0e-8;

const real PI = 2.0 * std::asin(1.0);

__global__ void initialize_boundaries(real* __restrict__ const a_new, real* __restrict__ const a,
                                      const real pi, const int nx, const int ny) {
    for (int iy = blockIdx.x * blockDim.x + threadIdx.x; iy < ny; iy += blockDim.x * gridDim.x) {
        const real y0 = sin(2.0 * pi * iy / (ny - 1));
        a[iy * nx + 0] = y0;
        a[iy * nx + (nx - 1)] = y0;
        a_new[iy * nx + 0] = y0;
        a_new[iy * nx + (nx - 1)] = y0;
    }
}

template <int BLOCK_DIM_X, int BLOCK_DIM_Y>
__global__ void jacobi_kernel(real* __restrict__ const a_new, const real* __restrict__ const a,
                              real* __restrict__ const l2_norm, const int iy_start,
                              const int iy_end, const int nx, const int ny,
                              const bool calculate_norm) {
#ifdef HAVE_CUB
    typedef hipcub::BlockReduce<real, BLOCK_DIM_X, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, BLOCK_DIM_Y>
        BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
#endif  // HAVE_CUB
    int iy = blockIdx.y * blockDim.y + threadIdx.y + iy_start;
    int ix = blockIdx.x * blockDim.x + threadIdx.x + 1;
    real local_l2_norm = 0.0;

    if (iy < iy_end && ix < (nx - 1)) {
        const real new_val = 0.25 * (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] +
                                     a[(iy + 1) * nx + ix] + a[(iy - 1) * nx + ix]);
        a_new[iy * nx + ix] = new_val;
        if (1 == iy) {
            a_new[(ny - 1) * nx + ix] = new_val;
        }
        if ((ny - 2) == iy) {
            a_new[0 * nx + ix] = new_val;
        }

        if (calculate_norm) {
            real residue = new_val - a[iy * nx + ix];
            local_l2_norm += residue * residue;
        }
    }
    if (calculate_norm) {
#ifdef HAVE_CUB
        real block_l2_norm = BlockReduce(temp_storage).Sum(local_l2_norm);
        if (0 == threadIdx.y && 0 == threadIdx.x) atomicAdd(l2_norm, block_l2_norm);
#else
        atomicAdd(l2_norm, local_l2_norm);
#endif  // HAVE_CUB
    }
}

double single_gpu(const int nx, const int ny, const int iter_max, real* const a_ref,
                  const int nccheck, const bool print);

template <typename T>
T get_argval(char** begin, char** end, const std::string& arg, const T default_val) {
    T argval = default_val;
    char** itr = std::find(begin, end, arg);
    if (itr != end && ++itr != end) {
        std::istringstream inbuf(*itr);
        inbuf >> argval;
    }
    return argval;
}

bool get_arg(char** begin, char** end, const std::string& arg) {
    char** itr = std::find(begin, end, arg);
    if (itr != end) {
        return true;
    }
    return false;
}

int main(int argc, char* argv[]) {
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 1000);
    const int nccheck = get_argval<int>(argv, argv + argc, "-nccheck", 1);
    const int nx = get_argval<int>(argv, argv + argc, "-nx", 16384);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", 16384);
    const bool csv = get_arg(argv, argv + argc, "-csv");

    real* a;
    real* a_new;

    real* a_ref;
    double runtime_serial = 0.0;

    CUDA_RT_CALL(hipSetDevice(0));
    CUDA_RT_CALL(hipFree(0));

    CUDA_RT_CALL(hipMallocManaged(&a_ref, nx * ny * sizeof(real)));
    runtime_serial = single_gpu(nx, ny, iter_max, a_ref, nccheck, !csv);

    CUDA_RT_CALL(hipMallocManaged(&a, nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMallocManaged(&a_new, nx * ny * sizeof(real)));

    CUDA_RT_CALL(hipMemset(a, 0, nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMemset(a_new, 0, nx * ny * sizeof(real)));

    // Set diriclet boundary conditions on left and right boarder
    initialize_boundaries<<<ny / 128 + 1, 128>>>(a, a_new, PI, nx, ny);
    CUDA_RT_CALL(hipGetLastError());
    CUDA_RT_CALL(hipDeviceSynchronize());

    if (!csv)
        printf(
            "Jacobi relaxation: %d iterations on %d x %d mesh with norm check "
            "every %d iterations\n",
            iter_max, ny, nx, nccheck);

    real l2_norm = 1.0;

    hipEvent_t compute_done[2][MAX_NUM_DEVICES];

    bool result_correct = true;
    int num_devices = 0;
    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));
#pragma omp parallel num_threads(num_devices) firstprivate(a, a_new)
    {
        int dev_id = omp_get_thread_num();

        CUDA_RT_CALL(hipSetDevice(dev_id));
        CUDA_RT_CALL(hipFree(0));

        // ny - 2 rows are distributed amongst `size` ranks in such a way
        // that each rank gets either (ny - 2) / size or (ny - 2) / size + 1 rows.
        // This optimizes load balancing when (ny - 2) % size != 0
        int chunk_size;
        int chunk_size_low = ny / num_devices;
        int chunk_size_high = chunk_size_low + 1;
        // To calculate the number of ranks that need to compute an extra row,
        // the following formula is derived from this equation:
        // num_ranks_low * chunk_size_low + (size - num_ranks_low) * (chunk_size_low + 1) = ny - 2
        int num_ranks_low = num_devices * chunk_size_low + num_devices -
                            ny;  // Number of ranks with chunk_size = chunk_size_low
        if (dev_id < num_ranks_low)
            chunk_size = chunk_size_low;
        else
            chunk_size = chunk_size_high;

        // Calculate local domain boundaries
        int iy_start;
        if (dev_id < num_ranks_low) {
            iy_start = dev_id * chunk_size_low;
        } else {
            iy_start = num_ranks_low * chunk_size_low + (dev_id - num_ranks_low) * chunk_size_high;
        }
        int iy_end = iy_start + chunk_size;
        // Do not process boundaries
        iy_start = std::max(iy_start, 1);
        iy_end = std::min(iy_end, ny - 1);

        const int top = dev_id > 0 ? dev_id - 1 : (num_devices - 1);
        int canAccessPeer = 0;
        CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, top));
        if (canAccessPeer) {
            CUDA_RT_CALL(hipDeviceEnablePeerAccess(top, 0));
        }
        const int bottom = (dev_id + 1) % num_devices;
        canAccessPeer = 0;
        CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, bottom));
        if (top != bottom && canAccessPeer) {
            CUDA_RT_CALL(hipDeviceEnablePeerAccess(bottom, 0));
        }

#ifdef UM_HINTS
        CUDA_RT_CALL(hipMemAdvise(a + iy_start * nx, (iy_end - iy_start) * nx * sizeof(real),
                                   hipMemAdviseSetPreferredLocation, dev_id));
        CUDA_RT_CALL(hipMemAdvise(a + (iy_start - 1) * nx, nx * sizeof(real),
                                   hipMemAdviseSetAccessedBy, dev_id));
        CUDA_RT_CALL(
            hipMemAdvise(a + iy_end * nx, nx * sizeof(real), hipMemAdviseSetAccessedBy, dev_id));
        CUDA_RT_CALL(hipMemAdvise(a_new + iy_start * nx, (iy_end - iy_start) * nx * sizeof(real),
                                   hipMemAdviseSetPreferredLocation, dev_id));
        CUDA_RT_CALL(hipMemAdvise(a_new + (iy_start - 1) * nx, nx * sizeof(real),
                                   hipMemAdviseSetAccessedBy, dev_id));
        CUDA_RT_CALL(hipMemAdvise(a_new + iy_end * nx, nx * sizeof(real),
                                   hipMemAdviseSetAccessedBy, dev_id));
#endif  // UM_HINTS

        constexpr int dim_block_x = 32;
        constexpr int dim_block_y = 32;
        dim3 dim_grid((nx + dim_block_x - 1) / dim_block_x,
                      (ny + (num_devices * dim_block_y) - 1) / (num_devices * dim_block_y), 1);

        real* l2_norm_d;
        real* l2_norm_h;

        CUDA_RT_CALL(hipMalloc(&l2_norm_d, sizeof(real)));
        CUDA_RT_CALL(hipHostMalloc(&l2_norm_h, sizeof(real)));

        CUDA_RT_CALL(hipEventCreateWithFlags(compute_done[0] + dev_id, hipEventDisableTiming));
        CUDA_RT_CALL(hipEventCreateWithFlags(compute_done[1] + dev_id, hipEventDisableTiming));

        CUDA_RT_CALL(hipDeviceSynchronize());

        int iter = 0;
        bool calculate_norm = true;
#pragma omp master
        { l2_norm = 1.0; }

#pragma omp barrier
        double start = omp_get_wtime();
        PUSH_RANGE("Jacobi solve", 0)
        while (l2_norm > tol && iter < iter_max) {
            CUDA_RT_CALL(hipMemsetAsync(l2_norm_d, 0, sizeof(real), 0));

// need to wait for other threads due to sharing of a, a_new and compute_done
// between threads
#pragma omp barrier
            CUDA_RT_CALL(hipStreamWaitEvent(0, compute_done[iter % 2][top], 0));
            CUDA_RT_CALL(hipStreamWaitEvent(0, compute_done[iter % 2][bottom], 0));

            calculate_norm = (iter % nccheck) == 0 || (!csv && (iter % 100) == 0);
            jacobi_kernel<dim_block_x, dim_block_y><<<dim_grid, {dim_block_x, dim_block_y, 1}>>>(
                a_new, a, l2_norm_d, iy_start, iy_end, nx, ny, calculate_norm);
            CUDA_RT_CALL(hipGetLastError());
            CUDA_RT_CALL(hipEventRecord(compute_done[(iter + 1) % 2][dev_id], 0));
#pragma omp barrier

            if (calculate_norm) {
                CUDA_RT_CALL(
                    hipMemcpyAsync(l2_norm_h, l2_norm_d, sizeof(real), hipMemcpyDeviceToHost, 0));
#pragma omp barrier
#pragma omp single
                { l2_norm = 0.0; }
#pragma omp barrier
                CUDA_RT_CALL(hipStreamSynchronize(0));
#pragma omp atomic
                l2_norm += *(l2_norm_h);
#pragma omp barrier
#pragma omp single
                { l2_norm = std::sqrt(l2_norm); }
#pragma omp barrier
                if (!csv && (iter % 100) == 0) {
#pragma omp master
                    printf("%5d, %0.6f\n", iter, l2_norm);
                }
            }

            std::swap(a_new, a);
            iter++;
        }
        CUDA_RT_CALL(hipDeviceSynchronize());
#pragma omp barrier
        POP_RANGE
        double stop = omp_get_wtime();

#pragma omp barrier

#pragma omp master
        {
            result_correct = true;
            for (int iy = 1; result_correct && (iy < (ny - 1)); ++iy) {
                for (int ix = 1; result_correct && (ix < (nx - 1)); ++ix) {
                    if (std::fabs(a_ref[iy * nx + ix] - a[iy * nx + ix]) > tol) {
                        fprintf(stderr,
                                "ERROR: a[%d * %d + %d] = %f does not match %f "
                                "(reference)\n",
                                iy, nx, ix, a[iy * nx + ix], a_ref[iy * nx + ix]);
                        result_correct = false;
                    }
                }
            }
            if (result_correct) {
                if (csv) {
                    printf("multi_threaded_um, %d, %d, %d, %d, %d, 1, %f, %f\n", nx, ny, iter_max,
                           nccheck, num_devices, (stop - start), runtime_serial);
                } else {
                    printf("Num GPUs: %d.\n", num_devices);
                    printf(
                        "%dx%d: 1 GPU: %8.4f s, %d GPUs: %8.4f s, speedup: "
                        "%8.2f, "
                        "efficiency: %8.2f \n",
                        ny, nx, runtime_serial, num_devices, (stop - start),
                        runtime_serial / (stop - start),
                        runtime_serial / (num_devices * (stop - start)) * 100);
                }
            }
        }

        CUDA_RT_CALL(hipEventDestroy(compute_done[1][dev_id]));
        CUDA_RT_CALL(hipEventDestroy(compute_done[0][dev_id]));

        CUDA_RT_CALL(hipHostFree(l2_norm_h));
        CUDA_RT_CALL(hipFree(l2_norm_d));
        CUDA_RT_CALL(hipDeviceSynchronize());
    }

    CUDA_RT_CALL(hipFree(a_new));
    CUDA_RT_CALL(hipFree(a));

    CUDA_RT_CALL(hipFree(a_ref));

    return result_correct ? 0 : 1;
}

template <int BLOCK_DIM_X, int BLOCK_DIM_Y>
__global__ void jacobi_kernel(real* __restrict__ const a_new, const real* __restrict__ const a,
                              real* __restrict__ const l2_norm, const int iy_start,
                              const int iy_end, const int nx, const bool calculate_norm) {
#ifdef HAVE_CUB
    typedef hipcub::BlockReduce<real, BLOCK_DIM_X, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, BLOCK_DIM_Y>
        BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
#endif  // HAVE_CUB
    int iy = blockIdx.y * blockDim.y + threadIdx.y + iy_start;
    int ix = blockIdx.x * blockDim.x + threadIdx.x + 1;
    real local_l2_norm = 0.0;

    if (iy < iy_end && ix < (nx - 1)) {
        const real new_val = 0.25 * (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] +
                                     a[(iy + 1) * nx + ix] + a[(iy - 1) * nx + ix]);
        a_new[iy * nx + ix] = new_val;

        if (calculate_norm) {
            real residue = new_val - a[iy * nx + ix];
            local_l2_norm += residue * residue;
        }
    }
    if (calculate_norm) {
#ifdef HAVE_CUB
        real block_l2_norm = BlockReduce(temp_storage).Sum(local_l2_norm);
        if (0 == threadIdx.y && 0 == threadIdx.x) atomicAdd(l2_norm, block_l2_norm);
#else
        atomicAdd(l2_norm, local_l2_norm);
#endif  // HAVE_CUB
    }
}

double single_gpu(const int nx, const int ny, const int iter_max, real* const a_ref,
                  const int nccheck, const bool print) {
    real* a;
    real* a_new;

    hipStream_t compute_stream;
    hipStream_t push_top_stream;
    hipStream_t push_bottom_stream;
    hipEvent_t compute_done;
    hipEvent_t push_top_done;
    hipEvent_t push_bottom_done;

    real* l2_norm_d;
    real* l2_norm_h;

    int iy_start = 1;
    int iy_end = (ny - 1);

    CUDA_RT_CALL(hipMalloc(&a, nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMalloc(&a_new, nx * ny * sizeof(real)));

    CUDA_RT_CALL(hipMemset(a, 0, nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMemset(a_new, 0, nx * ny * sizeof(real)));

    // Set diriclet boundary conditions on left and right boarder
    initialize_boundaries<<<ny / 128 + 1, 128>>>(a, a_new, PI, nx, ny);
    CUDA_RT_CALL(hipGetLastError());
    CUDA_RT_CALL(hipDeviceSynchronize());

    CUDA_RT_CALL(hipStreamCreate(&compute_stream));
    CUDA_RT_CALL(hipStreamCreate(&push_top_stream));
    CUDA_RT_CALL(hipStreamCreate(&push_bottom_stream));
    CUDA_RT_CALL(hipEventCreateWithFlags(&compute_done, hipEventDisableTiming));
    CUDA_RT_CALL(hipEventCreateWithFlags(&push_top_done, hipEventDisableTiming));
    CUDA_RT_CALL(hipEventCreateWithFlags(&push_bottom_done, hipEventDisableTiming));

    CUDA_RT_CALL(hipMalloc(&l2_norm_d, sizeof(real)));
    CUDA_RT_CALL(hipHostMalloc(&l2_norm_h, sizeof(real)));

    CUDA_RT_CALL(hipDeviceSynchronize());

    if (print)
        printf(
            "Single GPU jacobi relaxation: %d iterations on %d x %d mesh with "
            "norm "
            "check every %d iterations\n",
            iter_max, ny, nx, nccheck);

    constexpr int dim_block_x = 32;
    constexpr int dim_block_y = 32;
    dim3 dim_grid((nx + dim_block_x - 1) / dim_block_x, (ny + dim_block_y - 1) / dim_block_y, 1);

    int iter = 0;
    bool calculate_norm = true;
    real l2_norm = 1.0;

    double start = omp_get_wtime();
    PUSH_RANGE("Jacobi solve", 0)
    while (l2_norm > tol && iter < iter_max) {
        CUDA_RT_CALL(hipMemsetAsync(l2_norm_d, 0, sizeof(real), compute_stream));

        CUDA_RT_CALL(hipStreamWaitEvent(compute_stream, push_top_done, 0));
        CUDA_RT_CALL(hipStreamWaitEvent(compute_stream, push_bottom_done, 0));

        calculate_norm = (iter % nccheck) == 0 || (print && ((iter % 100) == 0));

        jacobi_kernel<dim_block_x, dim_block_y>
            <<<dim_grid, {dim_block_x, dim_block_y, 1}, 0, compute_stream>>>(
                a_new, a, l2_norm_d, iy_start, iy_end, nx, calculate_norm);
        CUDA_RT_CALL(hipGetLastError());
        CUDA_RT_CALL(hipEventRecord(compute_done, compute_stream));

        if (calculate_norm) {
            CUDA_RT_CALL(hipMemcpyAsync(l2_norm_h, l2_norm_d, sizeof(real), hipMemcpyDeviceToHost,
                                         compute_stream));
        }

        // Apply periodic boundary conditions

        CUDA_RT_CALL(hipStreamWaitEvent(push_top_stream, compute_done, 0));
        CUDA_RT_CALL(hipMemcpyAsync(a_new, a_new + (iy_end - 1) * nx, nx * sizeof(real),
                                     hipMemcpyDeviceToDevice, push_top_stream));
        CUDA_RT_CALL(hipEventRecord(push_top_done, push_top_stream));

        CUDA_RT_CALL(hipStreamWaitEvent(push_bottom_stream, compute_done, 0));
        CUDA_RT_CALL(hipMemcpyAsync(a_new + iy_end * nx, a_new + iy_start * nx, nx * sizeof(real),
                                     hipMemcpyDeviceToDevice, compute_stream));
        CUDA_RT_CALL(hipEventRecord(push_bottom_done, push_bottom_stream));

        if (calculate_norm) {
            CUDA_RT_CALL(hipStreamSynchronize(compute_stream));
            l2_norm = *l2_norm_h;
            l2_norm = std::sqrt(l2_norm);
            if (print && (iter % 100) == 0) printf("%5d, %0.6f\n", iter, l2_norm);
        }

        std::swap(a_new, a);
        iter++;
    }
    POP_RANGE
    double stop = omp_get_wtime();

    CUDA_RT_CALL(hipMemcpy(a_ref, a, nx * ny * sizeof(real), hipMemcpyDeviceToHost));

    CUDA_RT_CALL(hipEventDestroy(push_bottom_done));
    CUDA_RT_CALL(hipEventDestroy(push_top_done));
    CUDA_RT_CALL(hipEventDestroy(compute_done));
    CUDA_RT_CALL(hipStreamDestroy(push_bottom_stream));
    CUDA_RT_CALL(hipStreamDestroy(push_top_stream));
    CUDA_RT_CALL(hipStreamDestroy(compute_stream));

    CUDA_RT_CALL(hipHostFree(l2_norm_h));
    CUDA_RT_CALL(hipFree(l2_norm_d));

    CUDA_RT_CALL(hipFree(a_new));
    CUDA_RT_CALL(hipFree(a));
    return (stop - start);
}
