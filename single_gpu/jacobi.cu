#include "hip/hip_runtime.h"
/* Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include <algorithm>
#include <array>
#include <climits>
#include <cmath>
#include <cstdio>
#include <iostream>
#include <iterator>
#include <sstream>
#include <cstdlib>

#include <omp.h>

#ifdef HAVE_CUB
#include <cub/block/block_reduce.cuh>
#endif  // HAVE_CUB

#ifdef USE_NVTX
#include <nvtx3/nvToolsExt.h>

const uint32_t colors[] = {0x0000ff00, 0x000000ff, 0x00ffff00, 0x00ff00ff,
                           0x0000ffff, 0x00ff0000, 0x00ffffff};
const int num_colors = sizeof(colors) / sizeof(uint32_t);

#define PUSH_RANGE(name, cid)                              \
    {                                                      \
        int color_id = cid;                                \
        color_id = color_id % num_colors;                  \
        nvtxEventAttributes_t eventAttrib = {0};           \
        eventAttrib.version = NVTX_VERSION;                \
        eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;  \
        eventAttrib.colorType = NVTX_COLOR_ARGB;           \
        eventAttrib.color = colors[color_id];              \
        eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
        eventAttrib.message.ascii = name;                  \
        nvtxRangePushEx(&eventAttrib);                     \
    }
#define POP_RANGE nvtxRangePop();
#else
#define PUSH_RANGE(name, cid)
#define POP_RANGE
#endif

#define CUDA_RT_CALL(call)                                                                  \
    {                                                                                       \
        hipError_t cudaStatus = call;                                                      \
        if (hipSuccess != cudaStatus) {                                                    \
            fprintf(stderr,                                                                 \
                    "ERROR: CUDA RT call \"%s\" in line %d of file %s failed "              \
                    "with "                                                                 \
                    "%s (%d).\n",                                                           \
                    #call, __LINE__, __FILE__, hipGetErrorString(cudaStatus), cudaStatus); \
            exit( cudaStatus );                                                             \
        }                                                                                   \
    }

typedef float real;
constexpr real tol = 1.0e-8;

const real PI = 2.0 * std::asin(1.0);

__global__ void initialize_boundaries(real* __restrict__ const a_new, real* __restrict__ const a,
                                      const real pi, const int nx, const int ny) {
    for (int iy = blockIdx.x * blockDim.x + threadIdx.x; iy < ny; iy += blockDim.x * gridDim.x) {
        const real y0 = sin(2.0 * pi * iy / (ny - 1));
        a[iy * nx + 0] = y0;
        a[iy * nx + (nx - 1)] = y0;
        a_new[iy * nx + 0] = y0;
        a_new[iy * nx + (nx - 1)] = y0;
    }
}

template <int BLOCK_DIM_X, int BLOCK_DIM_Y>
__global__ void jacobi_kernel(real* __restrict__ const a_new, const real* __restrict__ const a,
                              real* __restrict__ const l2_norm, const int iy_start,
                              const int iy_end, const int nx) {
#ifdef HAVE_CUB
    typedef hipcub::BlockReduce<real, BLOCK_DIM_X, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, BLOCK_DIM_Y>
        BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
#endif  // HAVE_CUB
    const int iy = blockIdx.y * blockDim.y + threadIdx.y + 1;
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    real local_l2_norm = 0.0;

    if (iy < iy_end) {
        if (ix >= 1 && ix < (nx - 1)) {
            const real new_val = 0.25 * (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] +
                                         a[(iy + 1) * nx + ix] + a[(iy - 1) * nx + ix]);
            a_new[iy * nx + ix] = new_val;

            // apply boundary conditions
            if (iy_start == iy) {
                a_new[iy_end * nx + ix] = new_val;
            }

            if ((iy_end - 1) == iy) {
                a_new[(iy_start - 1) * nx + ix] = new_val;
            }

            real residue = new_val - a[iy * nx + ix];
            local_l2_norm = residue * residue;
        }
    }
#ifdef HAVE_CUB
    real block_l2_norm = BlockReduce(temp_storage).Sum(local_l2_norm);
    if (0 == threadIdx.y && 0 == threadIdx.x) atomicAdd(l2_norm, block_l2_norm);
#else
    atomicAdd(l2_norm, local_l2_norm);
#endif  // HAVE_CUB
}

double noopt(const int nx, const int ny, const int iter_max, real* const a_ref_h, const int nccheck,
             const bool print);

template <typename T>
T get_argval(char** begin, char** end, const std::string& arg, const T default_val) {
    T argval = default_val;
    char** itr = std::find(begin, end, arg);
    if (itr != end && ++itr != end) {
        std::istringstream inbuf(*itr);
        inbuf >> argval;
    }
    return argval;
}

bool get_arg(char** begin, char** end, const std::string& arg) {
    char** itr = std::find(begin, end, arg);
    if (itr != end) {
        return true;
    }
    return false;
}

struct l2_norm_buf {
    hipEvent_t copy_done;
    real* d;
    real* h;
};

int main(int argc, char* argv[]) {
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 1000);
    const int nccheck = get_argval<int>(argv, argv + argc, "-nccheck", 1);
    const int nx = get_argval<int>(argv, argv + argc, "-nx", 16384);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", 16384);
    const bool csv = get_arg(argv, argv + argc, "-csv");

    if (nccheck != 1) {
        fprintf(stderr, "Only nccheck = 1 is supported\n");
        return -1;
    }

    real* a;
    real* a_new;

    hipStream_t compute_stream;
    hipStream_t copy_l2_norm_stream;
    hipStream_t reset_l2_norm_stream;

    hipEvent_t compute_done;
    hipEvent_t reset_l2_norm_done[2];

    real l2_norms[2];
    l2_norm_buf l2_norm_bufs[2];

    int iy_start = 1;
    int iy_end = (ny - 1);

    CUDA_RT_CALL(hipSetDevice(0));
    CUDA_RT_CALL(hipFree(0));

    CUDA_RT_CALL(hipMalloc(&a, nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMalloc(&a_new, nx * ny * sizeof(real)));

    CUDA_RT_CALL(hipMemset(a, 0, nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMemset(a_new, 0, nx * ny * sizeof(real)));

    // Set diriclet boundary conditions on left and right boarder
    initialize_boundaries<<<ny / 128 + 1, 128>>>(a, a_new, PI, nx, ny);
    CUDA_RT_CALL(hipGetLastError());
    CUDA_RT_CALL(hipDeviceSynchronize());

    CUDA_RT_CALL(hipStreamCreate(&compute_stream));
    CUDA_RT_CALL(hipStreamCreate(&copy_l2_norm_stream));
    CUDA_RT_CALL(hipStreamCreate(&reset_l2_norm_stream));
    CUDA_RT_CALL(hipEventCreateWithFlags(&compute_done, hipEventDisableTiming));
    CUDA_RT_CALL(hipEventCreateWithFlags(&reset_l2_norm_done[0], hipEventDisableTiming));
    CUDA_RT_CALL(hipEventCreateWithFlags(&reset_l2_norm_done[1], hipEventDisableTiming));

    for (int i = 0; i < 2; ++i) {
        CUDA_RT_CALL(hipEventCreateWithFlags(&l2_norm_bufs[i].copy_done, hipEventDisableTiming));
        CUDA_RT_CALL(hipMalloc(&l2_norm_bufs[i].d, sizeof(real)));
        CUDA_RT_CALL(hipMemset(l2_norm_bufs[i].d, 0, sizeof(real)));
        CUDA_RT_CALL(hipHostMalloc(&l2_norm_bufs[i].h, sizeof(real)));
        (*l2_norm_bufs[i].h) = 1.0;
    }

    CUDA_RT_CALL(hipDeviceSynchronize());

    if (!csv)
        printf(
            "Jacobi relaxation: %d iterations on %d x %d mesh with norm check "
            "every %d iterations\n",
            iter_max, ny, nx, nccheck);

    constexpr int dim_block_x = 32;
    constexpr int dim_block_y = 32;
    dim3 dim_grid((nx + dim_block_x - 1) / dim_block_x, (ny + dim_block_y - 1) / dim_block_y, 1);

    int iter = 0;
    for (int i = 0; i < 2; ++i) {
        l2_norms[i] = 0.0;
    }

    double start = omp_get_wtime();

    PUSH_RANGE("Jacobi solve", 0)

    bool l2_norm_greater_than_tol = true;
    while (l2_norm_greater_than_tol && iter < iter_max) {
        // on new iteration: old current vars are now previous vars, old
        // previous vars are no longer needed
        int prev = iter % 2;
        int curr = (iter + 1) % 2;

        // wait for memset from old previous iteration to complete
        CUDA_RT_CALL(hipStreamWaitEvent(compute_stream, reset_l2_norm_done[curr], 0));

        jacobi_kernel<dim_block_x, dim_block_y>
            <<<dim_grid, {dim_block_x, dim_block_y, 1}, 0, compute_stream>>>(
                a_new, a, l2_norm_bufs[curr].d, iy_start, iy_end, nx);
        CUDA_RT_CALL(hipGetLastError());
        CUDA_RT_CALL(hipEventRecord(compute_done, compute_stream));

        // perform L2 norm calculation
        if ((iter % nccheck) == 0 || (!csv && (iter % 100) == 0)) {
            CUDA_RT_CALL(hipStreamWaitEvent(copy_l2_norm_stream, compute_done, 0));
            CUDA_RT_CALL(hipMemcpyAsync(l2_norm_bufs[curr].h, l2_norm_bufs[curr].d, sizeof(real),
                                         hipMemcpyDeviceToHost, copy_l2_norm_stream));
            CUDA_RT_CALL(hipEventRecord(l2_norm_bufs[curr].copy_done, copy_l2_norm_stream));

            // make sure D2H copy is complete before using the data for
            // calculation
            CUDA_RT_CALL(hipEventSynchronize(l2_norm_bufs[prev].copy_done));

            l2_norms[prev] = *(l2_norm_bufs[prev].h);
            l2_norms[prev] = std::sqrt(l2_norms[prev]);
            l2_norm_greater_than_tol = (l2_norms[prev] > tol);

            if (!csv && (iter % 100) == 0) {
                printf("%5d, %0.6f\n", iter, l2_norms[prev]);
            }

            // reset everything for next iteration
            l2_norms[prev] = 0.0;
            *(l2_norm_bufs[prev].h) = 0.0;
            CUDA_RT_CALL(
                hipMemsetAsync(l2_norm_bufs[prev].d, 0, sizeof(real), reset_l2_norm_stream));
            CUDA_RT_CALL(hipEventRecord(reset_l2_norm_done[prev], reset_l2_norm_stream));
        }

        std::swap(a_new, a);
        iter++;
    }
    CUDA_RT_CALL(hipDeviceSynchronize());
    POP_RANGE
    double stop = omp_get_wtime();

    if (csv) {
        printf("single_gpu, %d, %d, %d, %d, %f\n", nx, ny, iter_max, nccheck, (stop - start));
    } else {
        printf("%dx%d: 1 GPU: %8.4f s\n", ny, nx, (stop - start));
    }

    for (int i = 0; i < 2; ++i) {
        CUDA_RT_CALL(hipHostFree(l2_norm_bufs[i].h));
        CUDA_RT_CALL(hipFree(l2_norm_bufs[i].d));
        CUDA_RT_CALL(hipEventDestroy(l2_norm_bufs[i].copy_done));
    }

    CUDA_RT_CALL(hipEventDestroy(reset_l2_norm_done[1]));
    CUDA_RT_CALL(hipEventDestroy(reset_l2_norm_done[0]));
    CUDA_RT_CALL(hipEventDestroy(compute_done));

    CUDA_RT_CALL(hipStreamDestroy(reset_l2_norm_stream));
    CUDA_RT_CALL(hipStreamDestroy(copy_l2_norm_stream));
    CUDA_RT_CALL(hipStreamDestroy(compute_stream));

    CUDA_RT_CALL(hipFree(a_new));
    CUDA_RT_CALL(hipFree(a));

    return 0;
}
